#include "hip/hip_runtime.h"
/* The MIT License

   Copyright (c) 2011 Akiyama_Laboratory , Tokyo Institute of Technology.

   Permission is hereby granted, free of charge, to any person obtaining
   a copy of this software and associated documentation files (the
   "Software"), to deal in the Software without restriction, including
   without limitation the rights to use, copy, modify, merge, publish,
   distribute, sublicense, and/or sell copies of the Software, and to
   permit persons to whom the Software is furnished to do so, subject to
   the following conditions:

   The above copyright notice and this permission notice shall be 
   included in all copies or substantial portions of the Software.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
   EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
   MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
   NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
   BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
   ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
   CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE 
   SOFTWARE.
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil.h>
#include <stdint.h>
#include "common.h"
#include "aligner_gpu.h"

#define NUMBER_CONVERT_BLOCKS 128
#define NUMBER_CONVERT_THREADS 256

#define NUMBER_COUNT_BLOCKS 128
#define NUMBER_COUNT_THREADS 256

#define NUMBER_SET_BLOCKS 128
#define NUMBER_SET_THREADS 256

#define NUMBER_SCORE_BLOCKS 128
#define NUMBER_SCORE_THREADS 256

texture<int,  1, hipReadModeElementType> score_matrix_texture;
texture<uint8_t, 1, hipReadModeElementType> query_sequences_texture;

int g_device;
int *g_d_score_matrix;
uint32_t g_score_matrix_size;
uint8_t *g_d_query_sequences;
uint32_t g_query_sequences_size;
uint8_t *g_d_db_sequences;
uint32_t g_db_sequences_size;
uint32_t *g_d_keys_count;
uint32_t g_keys_count_size;
uint32_t *g_d_positions;
uint32_t g_positions_size;
uint32_t *g_d_keys;
uint32_t g_keys_size;
uint32_t *g_d_alignment_count_list;
uint32_t *g_alignment_count_list;
uint32_t g_alignment_count_list_size;
uint32_t g_start_query_id;
uint32_t g_query_count;
uint32_t *g_d_scores;
uint32_t g_scores_size;
uint32_t *g_d_starts;
uint32_t g_starts_size;
uint32_t *g_d_ends;
uint32_t g_ends_size;

__global__ void ConvertToKey
(
    uint8_t query_sequences[],
    uint32_t query_sequence_length,
    uint32_t number_query_sequences,
    uint32_t list_length,
    uint32_t seed,
    uint32_t seed_length,
    uint32_t shift_size,
    uint32_t keys[],
    uint32_t keys_length
)
{
  uint32_t i,j;
  uint32_t s;
  uint32_t key;
  uint32_t query_id;
  uint32_t query_offset;
  uint32_t list_id;
  uint32_t stride = gridDim.x*blockDim.x;;


  for (i = blockIdx.x*blockDim.x + threadIdx.x; i < keys_length; i += stride) {
    query_id = i/list_length;
    list_id = i%list_length;
    query_offset = (query_id*query_sequence_length) + (list_id*shift_size);
    for (j = 0, s = seed, key = 0; s != 0; ++j, s >>= 1) {
      if (s & 1) {
        key = key << CHARACTER_SIZE;
        key = key | query_sequences[query_offset + j];
      }
    }
    keys[i] = key;
  }

  return;
}

__global__ void CountQueryAlignment
(
    uint32_t number_query_sequences,
    uint32_t list_length,
    uint32_t keys[],
    uint32_t keys_count[],
    uint32_t positions[],
    uint32_t threshold,
    uint32_t shift_size,
    uint32_t log_region_size,
    uint32_t alignment_count_list[]
)
{
  uint32_t i,j,k,l;
  uint32_t d;
  uint32_t distance;
  uint32_t next_distance;
  uint32_t count;
  uint32_t next_count;
  uint32_t distance_list[MAX_LIST_SIZE];
  uint32_t positions_end_list[MAX_LIST_SIZE];
  uint32_t positions_id_list[MAX_LIST_SIZE];
  uint32_t count_list[MAX_LIST_SIZE];
  uint32_t sequence_offset;
  uint32_t number_alignment_list;
  uint32_t key;
  uint32_t keys_offset;
  uint32_t stride = gridDim.x*blockDim.x;


  // init
  for (i = 0; i < list_length; ++i) {
    distance_list[i] = UINT_MAX;
    positions_id_list[i] = 0;
    positions_end_list[i] = 0;
    count_list[i] = 0;
  }
  --threshold;

  for (i = blockIdx.x*blockDim.x + threadIdx.x; i < number_query_sequences; i += stride) {
    
    // init
    number_alignment_list = 0;
    keys_offset = i*list_length;
    for (j = 0; j < list_length; ++j) {
      sequence_offset = j*shift_size;
      key = keys[keys_offset + j];
      positions_id_list[j] = keys_count[key];
      positions_end_list[j] = keys_count[key + 1];

      for (k = positions_id_list[j]; k < positions_end_list[j] && positions[k] < sequence_offset; ++k)
        ;

      distance_list[j] = UINT_MAX;
      if (k < positions_end_list[j]) {
        distance_list[j] = (positions[k] - sequence_offset) >> log_region_size;
        ++k;
      }
      positions_id_list[j] = k;
      
    }

    distance = 0;
    count = 0;

    while (1) {
      // check min distance
      next_count = 1;
      next_distance = distance_list[0];
      count_list[0] = 0;
      for (j = 1; j < list_length; ++j) {
        if (distance_list[j] < next_distance) {
          next_distance = distance_list[j];
          next_count = 1;
          count_list[0] = j;
        } else if (distance_list[j] == next_distance) {
          count_list[next_count] = j;
          ++next_count;
        }
      }
      
      if (next_distance == UINT_MAX) {
        break;
      }
      
      // set distance list
      for (j = 0; j < next_count; ++j) {
        k = count_list[j];
        sequence_offset = k*shift_size;
        d = UINT_MAX;
        for (l = positions_id_list[k]; l < positions_end_list[k]; ++l) {
          d = (positions[l] - sequence_offset) >> log_region_size;
          if (d != next_distance) {
            ++l;
            break;
          }
	  d = UINT_MAX;
        }
	distance_list[k] = d;
	positions_id_list[k] = l;
      }

      if ((next_distance - distance) == 1) {
        count += next_count;
      }

      if (count > threshold) {
        ++number_alignment_list;
      }

      count = next_count;
      distance = next_distance;
      
    }
    
    // last check
    if (count > threshold) {
      ++number_alignment_list;
    }
    
    alignment_count_list[i] = number_alignment_list;
    
  }
  return;
}

__global__ void SetAlignmentList
(
    uint32_t start_query_id,
    uint32_t number_set_querys,
    uint32_t alignment_list[],
    uint32_t alignment_count_list[],
    uint32_t list_length,
    uint32_t keys[],
    uint32_t keys_count[],
    uint32_t positions[],
    uint32_t threshold,
    uint32_t shift_size,
    uint32_t log_region_size
)
{
  uint32_t i,j,k,l;
  uint32_t d;
  uint32_t distance;
  uint32_t next_distance;
  uint32_t count;
  uint32_t next_count;
  uint32_t distance_list[MAX_LIST_SIZE];
  uint32_t positions_end_list[MAX_LIST_SIZE];
  uint32_t positions_id_list[MAX_LIST_SIZE];
  uint32_t count_list[MAX_LIST_SIZE];
  uint32_t sequence_offset;
  uint32_t alignment_list_id;
  uint32_t alignment_list_end_id;
  uint32_t key;
  uint32_t keys_offset;
  uint32_t stride = gridDim.x*blockDim.x;


  // init
  for (i = 0; i < list_length; ++i) {
    distance_list[i] = UINT_MAX;
    positions_id_list[i] = 0;
    positions_end_list[i] = 0;
    count_list[i] = 0;
  }
  --threshold;

  for (i = blockIdx.x*blockDim.x + threadIdx.x; i < number_set_querys; i += stride) {
    
    // init
    alignment_list_id = alignment_count_list[i];
    alignment_list_end_id = alignment_count_list[i + 1];
    keys_offset = (start_query_id + i)*list_length;

    for (j = 0; j < list_length; ++j) {
      sequence_offset = j*shift_size;
      key = keys[keys_offset + j];
      positions_id_list[j] = keys_count[key];
      positions_end_list[j] = keys_count[key + 1];
      
      for (k = positions_id_list[j]; k < positions_end_list[j] && positions[k] < sequence_offset; ++k)
        ;
      
      distance_list[j] = UINT_MAX;
      if (k < positions_end_list[j]) {
        distance_list[j] = (positions[k] - sequence_offset) >> log_region_size;
        ++k;
      }
      positions_id_list[j] = k;
      
    }

    distance = 0;
    count = 0;
    while (alignment_list_id < alignment_list_end_id) {
      // check min distance
      next_count = 1;
      next_distance = distance_list[0];
      count_list[0] = 0;
      for (j = 1; j < list_length; ++j) {
        if (distance_list[j] < next_distance) {
          next_distance = distance_list[j];
          next_count = 1;
          count_list[0] = j;
        } else if (distance_list[j] == next_distance) {
          count_list[next_count] = j;
          ++next_count;
        }
      }
     
      // set distance list
      for (j = 0; j < next_count; ++j) {
        k = count_list[j];
        sequence_offset = k*shift_size;
        d = UINT_MAX;
        for (l = positions_id_list[k]; l < positions_end_list[k]; ++l) {
          d = (positions[l] - sequence_offset) >> log_region_size;
          if (d != next_distance) {
            ++l;
            break;
          }
	  d = UINT_MAX;
        }
	distance_list[k] = d;
	positions_id_list[k] = l;
      }
    

      if ((next_distance - distance) == 1) {
        count += next_count;
      }

      if (count > threshold) {
        alignment_list[alignment_list_id] = distance << log_region_size;
	++alignment_list_id;
      }

      count = next_count;
      distance = next_distance;
    }
  }
  return;
}

__global__ void CalculateScore
(
 uint8_t db_sequence[],
 uint32_t db_length, // db sequence length
 uint32_t query_sequence_length, // one query length
 uint32_t alignment_count_list[],
 uint32_t alignment_count_list_length,
 uint32_t number_alignment_list, // the number of alignment_list
 uint32_t scores[], // score array in alignment_list
 uint32_t starts[], // start array in alignment_list
 uint32_t ends[], // end array in alignment_list
 uint32_t start_query_id,
 uint32_t base_search_length,
 uint32_t offset,
 int open_gap,
 int extend_gap
)
{
  uint8_t db_character;
  uint32_t i, j, k;
  int score = 0;
  int local_score = 0; // score in cell
  int max_score = 0; // max score at the alignment
  uint32_t max_end = 0; // alignment end of max score at the alignment
  uint32_t alignment_count_id;
  int db_offset;
  int query_offset;
  int score_matrix_offset;
  uint32_t db_search_length;
  uint32_t query_search_length;
  int dp_column[MAX_COLUMN_LENGTH];
  int insertion_column[MAX_COLUMN_LENGTH];
  int deletion_score;
  int temp_score; // stored score of previous cell
  uint32_t stride = gridDim.x*blockDim.x;
  
  alignment_count_id = 0;
  alignment_count_list_length = alignment_count_list_length - 1;
  query_search_length = query_sequence_length + 1;

  
  for (i = blockIdx.x*blockDim.x + threadIdx.x; i< number_alignment_list; i += stride) {

    // init
    db_offset = starts[i] - offset;
    if (db_offset < 0) {
      db_offset = 0;
    }
    db_search_length = base_search_length;
    if (db_offset + db_search_length > db_length) {
      db_search_length = db_length - db_offset;
    }

    for (; alignment_count_id < alignment_count_list_length; ++alignment_count_id) {
     if (alignment_count_list[alignment_count_id]  <= i && i < alignment_count_list[alignment_count_id + 1]) {
       break;
     }
    }
    query_offset = (start_query_id + alignment_count_id)*query_sequence_length - 1;

    max_score = 0;
    for (j = 0; j < query_search_length; ++j) {
      dp_column[j] = 0;
      insertion_column[j] = 0;
    }


    // calculate score
    for (j = 0; j < db_search_length; ++j) {
      if ((db_character = db_sequence[db_offset + j]) != SEQUENCE_END) {
        score_matrix_offset = db_character*ALPHABET_SIZE;
        temp_score = 0;
        deletion_score = 0;
        for (k = 1; k < query_search_length; ++k) {
          local_score = 0;
          // match or mismatch
	  score = temp_score + tex1Dfetch(score_matrix_texture,
					  score_matrix_offset +
					  tex1Dfetch(query_sequences_texture, query_offset + k));
          if (score > local_score) {
            local_score = score;
          }

          // insertion
          if (insertion_column[k] + extend_gap < dp_column[k] + open_gap) {
            insertion_column[k] = dp_column[k] + open_gap;
          } else {
            insertion_column[k] += extend_gap;
          }

          if (insertion_column[k] > local_score) {
            local_score = insertion_column[k];
          }

          // deletion
          score = dp_column[k - 1] + open_gap;
          if (deletion_score + extend_gap < dp_column[k - 1] + open_gap) {
            deletion_score = dp_column[k - 1] + open_gap;
          } else {
            deletion_score += extend_gap;
          }

          if (deletion_score > local_score) {
            local_score = deletion_score;
          }

          // update score column
          temp_score = dp_column[k];
          dp_column[k] = local_score;

          // update max score
          if (local_score >= max_score) {
            max_score = local_score;
            max_end = j;
          }
        }

      } else {
        // reset score_column
        for (k = 0; k < query_search_length; ++k) {
          dp_column[k] = 0;
          insertion_column[k] = 0;
        }
      }

    }

    // set alignment result
    scores[i] = (uint32_t)max_score;
    ends[i] = db_offset + max_end;
    //ends[i] = query_offset;


  }
  return;
}

size_t GetNeededGPUMemorySize
(
  uint32_t seed,
  uint32_t shift_size,
  uint32_t max_list_length,
  uint32_t max_query_length,
  uint32_t max_number_queries,
  uint32_t max_db_length
)
{
  uint32_t seed_weight = 0;
  for (uint32_t s = seed; s > 0; s = s >> 1) {
  	if (s&1) {
  		++seed_weight;
  	}
  }
  uint32_t seed_length = 0;
  for (uint32_t s = seed; s != 0; s >>= 1, ++seed_length)
    ;

  // candidates
  size_t scores_size = sizeof(uint32_t)*max_list_length;
  size_t starts_size = sizeof(uint32_t)*max_list_length;
  size_t ends_size = sizeof(uint32_t)*max_list_length;
  
  // query
  size_t query_sequences_size = sizeof(uint8_t)*max_query_length;
  size_t alignment_count_list_size = sizeof(uint32_t)*(max_number_queries + 1);
  uint32_t keys_length = max_number_queries*((max_query_length - seed_length)/shift_size + 1);
  size_t key_size = sizeof(uint32_t)*keys_length;
  
  // db
  size_t db_sequence_size = sizeof(uint8_t)*max_db_length;
  uint32_t keys_count_length = (uint32_t)pow((double) ALPHABET_SIZE, (double) seed_weight) + 1;
  size_t keys_count_size = sizeof(uint32_t)*keys_count_length;
  size_t positions_size = sizeof(uint32_t)*max_db_length;
  
  // other
  size_t score_matrix_size = sizeof(int)*ALPHABET_SIZE*ALPHABET_SIZE;
  
  return scores_size +
  	starts_size +
  	ends_size +
  	query_sequences_size +
  	alignment_count_list_size +
  	key_size +
  	db_sequence_size +
  	keys_count_size +
  	positions_size +
	score_matrix_size;	
  	
}

// if global memory is over, return 1. otherwise return 0.
int CheckGpuMemory
(
   uint32_t seed,
   uint32_t shift_size,
   uint32_t max_list_length,
   uint32_t max_query_length,
   uint32_t max_number_queries,
   uint32_t max_db_length
)
{
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, g_device));
  size_t global_memory_size =  GetNeededGPUMemorySize(seed, shift_size, max_list_length, max_query_length, max_number_queries, max_db_length);

#if 0
    fprintf(stderr, "memory size.       %llu bytes\n", global_memory_size);
    fprintf(stderr, "global memory size.%llu bytes\n", deviceProp.totalGlobalMem);
    fflush(stderr);
#endif


  if (global_memory_size > deviceProp.totalGlobalMem) {
    return 1;
  } else {
    return 0;
  }
}

int InitGpu ()
{
  g_device = 0;
  g_d_score_matrix = NULL;
  g_score_matrix_size = 0;
  g_d_query_sequences = NULL;
  g_query_sequences_size = 0;
  g_d_db_sequences = NULL;
  g_db_sequences_size = 0;
  g_d_keys_count = NULL;
  g_keys_count_size = 0;
  g_d_positions = NULL;
  g_positions_size = 0;
  g_d_keys = NULL;
  g_keys_size = 0;
  g_d_alignment_count_list = NULL;
  g_alignment_count_list = NULL;
  g_alignment_count_list_size = 0;
  g_query_count = 0;
  g_d_scores = NULL;
  g_scores_size = 0;
  g_d_starts = NULL;
  g_starts_size = 0;
  g_d_ends = NULL;
  g_ends_size = 0;

  return 0;
}

int SetOptionGpu
(
 uint32_t max_list_length,
 int score_matrix[],
 int device
)
{
  g_device = device;
  CUDA_SAFE_CALL(hipSetDevice(g_device));

  // for > cuda 3.0 
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(ConvertToKey), hipFuncCachePreferL1);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(CountQueryAlignment), hipFuncCachePreferL1);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(SetAlignmentList), hipFuncCachePreferL1);
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(CalculateScore), hipFuncCachePreferL1);

  g_scores_size = sizeof(uint32_t)*max_list_length;
  g_starts_size = sizeof(uint32_t)*max_list_length;
  g_ends_size = sizeof(uint32_t)*max_list_length;

  CUDA_SAFE_CALL(hipMalloc((void **)&g_d_scores, g_scores_size));
  CUDA_SAFE_CALL(hipMalloc((void **)&g_d_starts, g_starts_size));
  CUDA_SAFE_CALL(hipMalloc((void **)&g_d_ends, g_ends_size));

  g_score_matrix_size = sizeof(int)*ALPHABET_SIZE*ALPHABET_SIZE;
  
  // set score matrix to device
  CUDA_SAFE_CALL(hipMalloc((void **)&g_d_score_matrix, g_score_matrix_size));
  CUDA_SAFE_CALL(hipMemcpy(g_d_score_matrix, score_matrix,
			    g_score_matrix_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTexture(NULL, score_matrix_texture, g_d_score_matrix, g_score_matrix_size));
  return 0;
}

void printGpuInfo(int device){
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  fprintf(stdout, "  [GPU] device: \"%s\"\n", deviceProp.name);
  fprintf(stdout, "  [GPU] global memory size: %u bytes (%gMB)\n", deviceProp.totalGlobalMem, (deviceProp.totalGlobalMem/1048576.0) );
  //fprintf(stdout, "  [GPU] number of cores: %d\n", nGpuArchCoresPerSM[deviceProp.major] * deviceProp.multiProcessorCount);
}

int SetQueryGpu
(
 uint8_t sequences[], 
 uint32_t number_sequences,
 uint32_t sequence_length
 )
{
  uint32_t new_size;
  new_size = sizeof(uint8_t)*number_sequences*sequence_length;
  if (new_size > g_query_sequences_size) {
    if (g_d_query_sequences != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_query_sequences));
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_query_sequences, new_size));
     g_query_sequences_size = new_size;
  }
  CUDA_SAFE_CALL(hipMemcpy(g_d_query_sequences, sequences,
			    new_size, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipBindTexture(NULL, query_sequences_texture, g_d_query_sequences, 
				 new_size));

  new_size = sizeof(uint32_t)*(number_sequences + 1);
  if (new_size > g_alignment_count_list_size) {
    if (g_d_alignment_count_list != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_alignment_count_list));
    }
    if (g_alignment_count_list != NULL) {
      free(g_alignment_count_list);
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_alignment_count_list, new_size));
    g_alignment_count_list= (uint32_t *)malloc(sizeof(uint32_t)*(number_sequences + 1));
    if (g_alignment_count_list == NULL) {
      return 1;
    }
    g_alignment_count_list_size = new_size;
  }
  
  return 0;
}

int SetDbGpu
(
 uint8_t sequences[],
 uint32_t sequences_legnth,
 uint32_t keys_count[],
 uint32_t keys_count_length,
 uint32_t positions[],
 uint32_t positions_length
 )
{
  uint32_t new_size = sizeof(uint8_t)*sequences_legnth;
  if (new_size > g_db_sequences_size) {
    if (g_d_db_sequences != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_db_sequences));
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_db_sequences, new_size));
    g_db_sequences_size = new_size;

  }
  CUDA_SAFE_CALL(hipMemcpy(g_d_db_sequences, sequences, new_size, hipMemcpyHostToDevice));
  
  new_size = sizeof(uint32_t)*keys_count_length;
  if (new_size > g_keys_count_size) {
    if (g_d_keys_count != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_keys_count));
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_keys_count, new_size));
    g_keys_count_size = new_size;
  }
  CUDA_SAFE_CALL(hipMemcpy(g_d_keys_count, keys_count, new_size, hipMemcpyHostToDevice));
  
  new_size = sizeof(uint32_t)*positions_length;
  if (new_size > g_positions_size) {
    if (g_d_positions != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_positions));
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_positions, new_size));
    g_positions_size = new_size;
  }
  CUDA_SAFE_CALL(hipMemcpy(g_d_positions, positions, new_size, hipMemcpyHostToDevice));

#if 0
    // debug ///////////////////////////////////////////////////
    fprintf(stderr, "keys_count\n");
    for (uint32_t i = 0; i < 10; ++i) {
      fprintf(stderr, "%u ", keys_count[i]);
    }
    fprintf(stderr, "\n");

    fprintf(stderr, "positions\n");
    for (uint32_t i = 0; i < 10; ++i) {
      fprintf(stderr, "%u ", positions[i]);
    }
    fprintf(stderr, "\n");
    /////////////////////////////////////////////////////////////
#endif
  
  return 0;
}

uint32_t SearchNextGpu
(
    uint32_t query_sequence_length,
    uint32_t number_query_sequences,
    uint32_t seed,
    uint32_t threshold,
    uint32_t shift_size,
    uint32_t log_region_size,
    uint32_t max_number_alignments,
    uint32_t start_query_id,
    uint32_t *alignment_count_list,
    uint32_t *starts
)
{
  uint32_t i,j;
  uint32_t s;
  uint32_t seed_length;
  uint32_t list_length;
  uint32_t keys_length;
  uint32_t query_count;
  uint32_t number_alignments;
  uint32_t new_size;

  // debug ////////////////////////////
  unsigned int timer;
  //float gpu_time;
  ///////////////////////////////////////

  if (start_query_id == number_query_sequences) {
    return 0;
  }
  g_start_query_id = start_query_id;

  //debug ////////////////////////////
  cutCreateTimer(&timer);
  ///////////////////////////////////////
  

  // set seed length
  for (seed_length = 0, s = seed; s != 0; s >>= 1, ++seed_length)
    ;
  
  list_length = (query_sequence_length - seed_length)/shift_size + 1;
  keys_length = number_query_sequences*list_length;


  new_size = sizeof(uint32_t)*keys_length;
  if (new_size > g_keys_size) {
    if (g_d_query_sequences != NULL) {
      CUDA_SAFE_CALL(hipFree(g_d_keys));
    }
    CUDA_SAFE_CALL(hipMalloc((void **)&g_d_keys, new_size));
    g_keys_size = new_size;
  }

  if (start_query_id == 0) { 
  //debug /////////////////////////
    cutResetTimer(timer);
    cutStartTimer(timer);
  ////////////////////////////////////


    ConvertToKey<<<NUMBER_CONVERT_BLOCKS, NUMBER_CONVERT_THREADS>>>(g_d_query_sequences, query_sequence_length, number_query_sequences, list_length, seed, seed_length, shift_size, g_d_keys, keys_length);
    
    // debug ////////////////////////////////////
    //hipDeviceSynchronize();
    //cutStopTimer(timer);
    //gpu_time = cutGetTimerValue(timer)*1.0e-03;
    //fprintf(stderr, "\n   convert into keys:  %9.3e [GPU]\n", gpu_time);
    /////////////////////////////////////////////
    
    
#if 0
    // debug ///////////////////////////////////////////////////
    fprintf(stderr, "keys\n");
    uint32_t *keys = (uint32_t *)malloc(sizeof(uint32_t)*keys_length);
    CUDA_SAFE_CALL(hipMemcpy(keys, g_d_keys,
			      sizeof(uint32_t)*keys_length, hipMemcpyDeviceToHost));
    for (i = 0; i < list_length; ++i) {
      fprintf(stderr, "%u ", keys[i]);
    }
    fprintf(stderr, "\n");
    free(keys);
    /////////////////////////////////////////////////////////////
#endif

#if 0
    // debug ///////////////////////////////////////////////////
    fprintf(stderr, "keys_count\n");
    uint32_t *keys_count = (uint32_t *)malloc(g_keys_count_size);
    CUDA_SAFE_CALL(hipMemcpy(keys_count, g_d_keys_count,
			      g_keys_count_size, hipMemcpyDeviceToHost));
    for (i = 0; i < 10; ++i) {
      fprintf(stderr, "%u ", keys_count[i]);
    }
    fprintf(stderr, "\n");
    free(keys_count);

    fprintf(stderr, "positions\n");
    uint32_t *positions = (uint32_t *)malloc(g_positions_size);
    CUDA_SAFE_CALL(hipMemcpy(positions, g_d_positions,
			      g_positions_size, hipMemcpyDeviceToHost));
    for (i = 0; i < 10; ++i) {
      fprintf(stderr, "%u ", positions[i]);
    }
    fprintf(stderr, "\n");
    free(positions);
    /////////////////////////////////////////////////////////////
#endif
    
    // count each query alignments
    alignment_count_list[0] = 0;
    //debug /////////////////////////
    cutResetTimer(timer);
    cutStartTimer(timer);
    /////////////////////////////////////
#if 0
    //debug /////////////////////////
    fprintf(stderr, "number_query_sequences %u \n", number_query_sequences);
    fprintf(stderr, "list_length %u \n", list_length);
    fprintf(stderr, "threshold %u \n", threshold);
    fprintf(stderr, "shift_size %u \n", shift_size);
    fprintf(stderr, "log_region_size %u \n", log_region_size);
    /////////////////////////////////////
#endif
    CountQueryAlignment<<<NUMBER_COUNT_BLOCKS, NUMBER_COUNT_THREADS>>>(number_query_sequences, list_length, 
								       g_d_keys, g_d_keys_count, g_d_positions,
								       threshold, shift_size, log_region_size,
								       g_d_alignment_count_list);

    // debug //////////////////
    //hipDeviceSynchronize();
    //cutStopTimer(timer);
    //gpu_time = (cutGetTimerValue(timer)*1.0e-03);
    //fprintf(stderr, "   count tickets: %9.3e [GPU]\n", gpu_time);
    ////////////////////////////
    g_alignment_count_list[0] = 0;
    CUDA_SAFE_CALL(hipMemcpy(&g_alignment_count_list[1], g_d_alignment_count_list,
			      sizeof(uint32_t)*number_query_sequences, hipMemcpyDeviceToHost));
    
#if 0
    // debug ///////////////////////////////////////////////////
    fprintf(stderr, "alignment count\n");
    for (i = 1; i <= /*number_query_sequences*/ 10; ++i) {
      fprintf(stderr, "%u ", alignment_count_list[i]);
      }
    fprintf(stderr, "\n");
    //return 0;
    /////////////////////////////////////////////////////////////
#endif
  }

  number_alignments = 0;
  alignment_count_list[0] = 0;
  for (i = start_query_id + 1, j = 1; 
       (number_alignments + g_alignment_count_list[i] < max_number_alignments) && 
	 (i <= number_query_sequences);
       ++i, ++j) {
    number_alignments += g_alignment_count_list[i];
    alignment_count_list[j] = number_alignments;
  }
  query_count = j - 1;

#if 0
  // debug ///////////////////////////////////////////////////
  fprintf(stderr, "start query id %u \n", start_query_id);
  fprintf(stderr, "query_count %u \n", query_count);
  fprintf(stderr, "alignment_count_list\n");
  for (i = 0; i < query_count + 1; ++i) {
    fprintf(stderr, "%u ", alignment_count_list[i]);
  }
  fprintf(stderr, "\n");
  /////////////////////////////////////////////////////////////
#endif

  CUDA_SAFE_CALL(hipMemcpy(g_d_alignment_count_list, alignment_count_list, 
			    sizeof(uint32_t)*(query_count + 1), hipMemcpyHostToDevice));
    

    //debug /////////////////////////
  cutResetTimer(timer);
  cutStartTimer(timer);
  ////////////////////////////////////
  
  SetAlignmentList<<<NUMBER_SET_BLOCKS, NUMBER_SET_THREADS>>>(start_query_id, query_count, g_d_starts, 
							      g_d_alignment_count_list, list_length, g_d_keys,
							      g_d_keys_count, g_d_positions, threshold, 
							      shift_size, log_region_size);

  // debug ////////////////////////////////////
  //hipDeviceSynchronize();
  //cutStopTimer(timer);
  //gpu_time = cutGetTimerValue(timer)*1.0e-03;
  //printf("   set tickets: %9.3e [GPU]\n", gpu_time);
  /////////////////////////////////////////////
  
  CUDA_SAFE_CALL(hipMemcpy(starts, g_d_starts, sizeof(uint32_t)*number_alignments, hipMemcpyDeviceToHost));
  
#if 0
  // debug ///////////////////////////////////////////////////
  fprintf(stderr, "starts\n");
  for (i = number_alignments - 10; i < number_alignments; ++i) {
  fprintf(stderr, "%u ", starts[i]);
  }
  fprintf(stderr, "\n");
  /////////////////////////////////////////////////////////////
#endif

  g_query_count = query_count;
  return query_count;
}

void  CalculateScoreGpu
(
 uint32_t db_length, // db sequence length
 uint32_t query_sequence_length, // one query length
 uint32_t number_alignment_list, // the number of alignment_list
 uint32_t scores[], // score array in alignment_list
 uint32_t ends[], // end array in alignment_list
 uint32_t base_search_length,
 uint32_t offset,
 int open_gap,
 int extend_gap
)
{
  dim3 dim_grid(NUMBER_SCORE_BLOCKS);
  dim3 dim_block(NUMBER_SCORE_THREADS);
#if 0
    // debug ///////////////////////////////////////////////////
    fprintf(stderr, "alignment_count_list_length\n");
    uint32_t alignment_count_list_length = g_query_count + 1;
    uint32_t *alignment_count_list = (uint32_t *)malloc(sizeof(uint32_t)*alignment_count_list_length);
    CUDA_SAFE_CALL(hipMemcpy(alignment_count_list, g_d_alignment_count_list,
			      sizeof(uint32_t)*alignment_count_list_length, hipMemcpyDeviceToHost));
    for (uint32_t i = 0; i < alignment_count_list_length; ++i) {
      fprintf(stderr, "%u ", alignment_count_list[i]);
    }
    fprintf(stderr, "\n");
    free(alignment_count_list);
    /////////////////////////////////////////////////////////////
#endif

#if 0
    //debug /////////////////////////
    fprintf(stderr, "query_sequence_length %u \n", query_sequence_length);
    fprintf(stderr, "alignment_count_list_length %u \n", g_query_count + 1);
    fprintf(stderr, "number_alignment_list %u \n", number_alignment_list);
    fprintf(stderr, "start_query_id %u \n", g_start_query_id);
    fprintf(stderr, "base_search_length %u \n", base_search_length);
    fprintf(stderr, "offset %u \n", offset);
    fprintf(stderr, "open_gap %d \n", open_gap);
    fprintf(stderr, "extend_gap %d \n", extend_gap);
    /////////////////////////////////////
#endif

  CalculateScore<<<dim_grid, dim_block>>>(g_d_db_sequences, db_length, query_sequence_length, 
					  g_d_alignment_count_list,
					  g_query_count + 1, number_alignment_list, 
					  g_d_scores, g_d_starts, g_d_ends, g_start_query_id, base_search_length, 
					  offset, open_gap, extend_gap);
  CUT_CHECK_ERROR("calculatealignment_listcore() execution failed.\n");

  // return score
  CUDA_SAFE_CALL(hipMemcpy(scores, g_d_scores, g_scores_size, hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(ends, g_d_ends, g_ends_size, hipMemcpyDeviceToHost));


  return;
}

int FreeGpu
(
 // no parameter
)
{
  // unbind texture
  CUDA_SAFE_CALL(hipUnbindTexture(score_matrix_texture));
  CUDA_SAFE_CALL(hipUnbindTexture(query_sequences_texture));

  // free gpu memory
  CUDA_SAFE_CALL(hipFree(g_d_db_sequences));
  CUDA_SAFE_CALL(hipFree(g_d_query_sequences));
  CUDA_SAFE_CALL(hipFree(g_d_score_matrix));
  CUDA_SAFE_CALL(hipFree(g_d_alignment_count_list));
  CUDA_SAFE_CALL(hipFree(g_d_scores));
  CUDA_SAFE_CALL(hipFree(g_d_starts));
  CUDA_SAFE_CALL(hipFree(g_d_ends));
 
  // free memory
  free(g_alignment_count_list);
  return 0;
}


